#include "hip/hip_runtime.h"
#ifndef _CUDA_TENSOR_CONV2_CU_
#define _CUDA_TENSOR_CONV2_CU_

#include "cuda_tensor.cuh"
#include "base/cuda_reduce.cuh"

namespace apex_tensor{

    namespace cuda_tensor{
        // suppport for kahan sum procedure
        namespace __conv2{
            inline __device__ void kahan_sum( float &sum, float &c_kahan, float val ){
                float y = val - c_kahan;
                float t = sum + y;
                c_kahan = ( t - sum ) -y;
                sum     = t; 
            }
            
            inline __device__ void add_sum( float &sum, float &c_kahan, float val ){
#ifdef __CUDA_CONV2_USE_KAHAN_SUM__
                kahan_sum( sum , c_kahan, val );
#else
                sum += val;            
#endif
            }
            
            inline __device__ void add_sum( float &sum, float val ){
                sum += val;
            }
            
#ifdef __CUDA_CONV2_USE_KAHAN_SUM__
            
#define  CUDA_CONV2_SUM_PARAM(x,y)   float &x, float &y
#define  CUDA_CONV2_SUM_VAR_DEF(x,y) float x = 0.0f, y = 0.0f
#define  CUDA_CONV2_SUM_ARG(x,y)     x, y
            
#else
            
#define  CUDA_CONV2_SUM_PARAM(x,y)   float &x
#define  CUDA_CONV2_SUM_VAR_DEF(x,y) float x = 0.0f
#define  CUDA_CONV2_SUM_ARG(x,y)     x
            
#endif            
        };

        // support for loading
        namespace __conv2{
            // load a segment into array, check whether the data is aligned 
            template< int x_size >
            inline __device__ void __load_line_shared_pad_rec_check_align( float m_shared[x_size],
                                                                           const __GT1D m_global,
                                                                           int x_start ){
                // noting: x_start may be mis-aligned
                const int x_shift = x_start & MEM_UNIT_MASK; // get the shifting area
                if( threadIdx.x >= x_shift ){
                    int cx = x_start + threadIdx.x - x_shift;
                    if( cx < m_global.x_max && cx >= 0 ){
                        m_shared[ threadIdx.x - x_shift ] = m_global[ cx ];
                    }else{
                        m_shared[ threadIdx.x - x_shift ] = 0.0f;
                    }
                }
                for( int x = MEM_UNIT ; x < x_size ; x += MEM_UNIT ){
                    int xx = x       + threadIdx.x - x_shift;
                    int cx = x_start + xx;
                    if( cx < m_global.x_max && cx >= 0 ){
                        m_shared[ xx ] = m_global[ cx ]; 
                    }else{
                        m_shared[ xx ] = 0.0f;
                    }
                }                
                if( threadIdx.x < x_shift ){
                    int xx = x_size  + threadIdx.x - x_shift;
                    int cx = x_start + xx;
                    if( cx < m_global.x_max && cx >= 0 ){
                        m_shared[ xx ] = m_global[ cx ];
                    }else{
                        m_shared[ xx ] = 0.0f;
                    }
                }                  
            }
            
            // load data into array, the x_start is ensured to be aligned 
            template< int x_size >
            inline __device__ void __load_line_shared_pad_rec_aligned( float m_shared[x_size],
                                                                       const __GT1D m_global,
                                                                       int x_start ){
                for( int x = 0 ; x < x_size ; x += MEM_UNIT ){
                    int xx = x       + threadIdx.x;
                    int cx = x_start + xx;
                    if( cx < m_global.x_max && cx >= 0 ){
                        m_shared[ xx ] = m_global[ cx ]; 
                    }else{
                        m_shared[ xx ] = 0.0f;
                    }
                }                
            }
            
            // load matrix into shared memory, dim_x = dim_y
            // pad exceeding dimsions with 0
            template<int y_size, int x_size,bool check_align>
            inline __device__ void __load_mat_shared_pad_rec( float m_shared[y_size][x_size], 
                                                              const __GT2D m_global, 
                                                              int y_start ,int x_start ){
                for( int y = 0; y < y_size; y += Y_UNIT ){
                    int yy =  y + threadIdx.y; // consant in warp
                    int cy =  y_start + yy;
                    if( cy < m_global.y_max && cy >= 0 ){
                        if( check_align ) 
                            __load_line_shared_pad_rec_check_align< x_size >( m_shared[ yy ] , m_global[ cy ] , x_start );
                        else
                            __load_line_shared_pad_rec_aligned< x_size >( m_shared[ yy ] , m_global[ cy ] , x_start );
                    }else{
                        for( int x = 0; x < x_size; x += MEM_UNIT )
                            m_shared[ yy ][ x + threadIdx.x ] = 0.0f;
                    }                                                                      
                }                  
            }
            
            // reverse load 
            template<int y_size,int x_size>
            inline __device__ void __load_mat_shared_reverse_rec( float m_shared[y_size][x_size], 
                                                                  const __GT2D g_filter ){
                for( int y = 0; y < y_size; y += Y_UNIT ){
                    for( int x = 0; x < x_size; x += MEM_UNIT ){                        
                        int yy =  y + threadIdx.y; // consant in warp
                        int xx =  x + threadIdx.x; // stride = 1
                        
                        if( yy < g_filter.y_max && xx < g_filter.x_max ){
                            // stride = 1 aligned
                            m_shared[ g_filter.y_max - yy - 1 ][ g_filter.x_max - xx - 1 ] = g_filter[ yy ][ xx ];
                        }
                    }
                }
            }  
        };
    };

    // conv2 valid 
    namespace cuda_tensor{
        /* 
           note: block_x and block_y are virtual block id,
           they may not equal to blockIdx.x and blockIdx.y
        */
        template<int y_size, int x_size>
        inline __device__ void __conv2_r_valid_procedure_rec( float &sum,
                                                              int block_y  , int block_x,                                    
                                                              float s_ft[y_size][x_size] ,
                                                              float s_mm[y_size+Y_UNIT][x_size+MEM_UNIT],
                                                              int ans_y_max, int ans_x_max,
                                                              const __GT2D mat,
                                                              const __GT2D filter ){
            // load filter into shared memory
            __conv2::__load_mat_shared_pad_rec<y_size, x_size, false>
                ( s_ft , filter, 0, 0 ); 
            // load matrix into shared memory
            __conv2::__load_mat_shared_pad_rec<y_size+Y_UNIT, x_size+MEM_UNIT, false>
                ( s_mm , mat, (block_y<<Y_UNIT_BITS), (block_x<<MEM_UNIT_BITS) );
            
            __syncthreads();
            
            const int y_idx = (block_y<<Y_UNIT_BITS) + threadIdx.y;
            const int x_idx = (block_x<<MEM_UNIT_BITS) + threadIdx.x;
            
            if( y_idx < ans_y_max && x_idx < ans_x_max  ){
                for( int dy = 0 ; dy < filter.y_max ; dy ++ ){
                    float ss = 0.0f;
                    for( int dx = 0 ; dx < filter.x_max ; dx ++ ){
                        // s_ft[dy,dx] get by broadcast
                        // s_mm get by stride = 1
                        ss += s_mm[ threadIdx.y + dy ][ threadIdx.x + dx ] * s_ft[ dy ][ dx ];
                    }
                    // for better accuracy
                    sum += ss;
                }
            }
        }
        
        template<int y_size, int x_size>
        inline __device__ void __conv2_r_valid_procedure_rec( float &sum,
                                                              int   h_idx,
                                                              int   block_y, int block_x, 
                                                              float s_ft[y_size][x_size],
                                                              float s_mm[y_size+Y_UNIT][x_size+MEM_UNIT],
                                                              int   ans_y_max, int ans_x_max,
                                                              const __GT3D mat,
                                                              const __GT4D filter ){        
            
            for( int v = 0 ;  v < filter.h_max ; v ++ ){                
                __conv2_r_valid_procedure_rec<y_size,x_size>
                    ( sum,
                      block_y   ,  block_x, 
                      s_ft, s_mm, 
                      ans_y_max ,  ans_x_max, 
                      mat[v] , filter[v][h_idx] );
                //wait for other threads
                __syncthreads(); 
            }
        }                       
        
        template<int st_m,int y_size,int x_size>
        __global__ void __conv2_r_valid_kernel_rec( int grid_width, 
                                                    __GT3D ans,                                                   
                                                    const __GT3D mat,
                                                    const __GT4D filter,
                                                    const __GT1D h_bias   ){
            // unzip the block index
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;

            __shared__ float bias;
            __shared__ float s_ft[y_size][x_size];
            __shared__ float s_mm[y_size+Y_UNIT][x_size+MEM_UNIT];
            
            // load the bias from data 
            if( threadIdx.y == Y_UNIT-1 && threadIdx.x == MEM_UNIT-1 ){
                // we use last thread to do the job, since
                // last thread may more likely to be idle
                bias = h_bias[ block_z ];
                // we don't sync threads here, note we may sync it in the latter operaton
            }

            float sum = 0.0f;
            
            __conv2_r_valid_procedure_rec<y_size,x_size>
                ( sum, block_z, block_y, block_x,
                  s_ft, s_mm, ans.y_max, ans.x_max, mat, filter );

            sum += bias;           
            
            const int  y_idx    = (block_y<<Y_UNIT_BITS) + threadIdx.y;
            const int  x_idx    = (block_x<<MEM_UNIT_BITS) + threadIdx.x;            
            if( y_idx < ans.y_max && x_idx < ans.x_max ){
                store_method::__store<st_m>( ans[ block_z ][ y_idx ][ x_idx ] , sum );    
            }   
        }
        
        template<int st_m>
        inline void conv2_r_valid_orign( GTensor3D &ans,
                                         const GTensor3D &mat,
                                         const GTensor4D &filter,
                                         const GTensor1D &h_bias ){
            // only 16,16 block is allowed to support maxpooling
            if( filter.y_max <= Y_UNIT && filter.x_max <= MEM_UNIT ){
                int  grid_height = (ans.y_max+Y_UNIT-1) >> Y_UNIT_BITS;
                int  grid_width  = (ans.x_max+MEM_UNIT-1) >> MEM_UNIT_BITS;           
                dim3 dimBlock( MEM_UNIT, Y_UNIT, 1 );
                dim3 dimGrid ( grid_width * grid_height ,  filter.z_max , 1 );
                __conv2_r_valid_kernel_rec <st_m,Y_UNIT,MEM_UNIT> <<<dimGrid,dimBlock>>> 
                    ( grid_width, __GT(ans) , __GT(mat), __GT(filter), __GT(h_bias) );
            }
            else{
                error("too large filter size");
            }
        }
    };

    // conv2_full
    namespace cuda_tensor{
        template<int y_size, int x_size>
        inline __device__ void __conv2_full_procedure_rec( float &sum,
                                                           int block_y, int block_x,
                                                           float s_ft[y_size   ][x_size],
                                                           float s_mm[y_size+Y_UNIT][x_size+MEM_UNIT],
                                                           int   ans_y_max, int ans_x_max,
                                                           const __GT2D mat,
                                                           const __GT2D filter ){
            // load filter into shared memory
            __conv2::__load_mat_shared_reverse_rec< y_size, x_size >( s_ft, filter );
            
            // load matrix into shared memory
            const int y_start = (block_y<<Y_UNIT_BITS) - filter.y_max + 1;
            const int x_start = (block_x<<MEM_UNIT_BITS) - filter.x_max + 1;
            __conv2::__load_mat_shared_pad_rec< y_size+16, x_size+MEM_UNIT, true>
                ( s_mm, mat, y_start, x_start );
            
            __syncthreads();
            
            const int y_idx = (block_y<<Y_UNIT_BITS) + threadIdx.y;
            const int x_idx = (block_x<<MEM_UNIT_BITS) + threadIdx.x;
            
            if( y_idx < ans_y_max && x_idx < ans_x_max ){
                for( int dy = 0; dy < filter.y_max; dy ++ ){
                    for( int dx = 0; dx < filter.x_max; dx ++ ){
                        // s_ft[dy,dx] get by broadcast
                        // s_mm get by stride = 1
                        sum += s_mm[ threadIdx.y + dy ][ threadIdx.x + dx ] * s_ft[ dy ][ dx ] ;
                    }
                }
            }
        }
        
        template<int y_size, int x_size>
        inline __device__ void __conv2_full_procedure_rec( float &sum,
                                                           int v_idx, int block_y, int block_x,
                                                           float s_ft[y_size   ][x_size] ,
                                                           float s_mm[y_size+Y_UNIT][x_size+MEM_UNIT] ,
                                                           const __GT3D ans,
                                                           const __GT3D mat,
                                                           const __GT4D filter ){        
            for( int h = 0 ; h < filter.z_max ; h ++ ){
                __conv2_full_procedure_rec<y_size,x_size>
                    ( sum ,
                      block_y, block_x,
                      s_ft , s_mm , ans.y_max, ans.x_max, mat[h] , filter[v_idx][h] ); 
                __syncthreads();
            }
        }

        /* convolution with bias */
        template<int st_m, int y_size, int x_size>
        __global__ void __conv2_full_kernel_rec( int grid_width,
                                                 __GT3D ans,
                                                 const __GT3D mat,
                                                 const __GT4D filter,
                                                 const __GT1D v_bias ){
            int block_z = blockIdx.y;
            int block_y = blockIdx.x / grid_width;
            int block_x = blockIdx.x % grid_width;
        
            __shared__ float bias;
            __shared__ float s_ft[y_size   ][x_size];
            __shared__ float s_mm[y_size+Y_UNIT][x_size+MEM_UNIT];

            //load the bias
            if( threadIdx.y == Y_UNIT-1 && threadIdx.x == MEM_UNIT-1 ){
                // we use last thread because last thread seems more likely to be idle
                // no need to sync because sync will occur in latter procedure
                bias = v_bias[ block_z ];
            }
        
            float sum = 0.0f;
            
            __conv2_full_procedure_rec<y_size,x_size>
                ( sum, block_z, block_y, block_x,
                  s_ft , s_mm , ans, mat, filter );        
            
            sum += bias;
                        
            const int  y_idx = (block_y<<Y_UNIT_BITS) + threadIdx.y;
            const int  x_idx = (block_x<<MEM_UNIT_BITS) + threadIdx.x;
            
            if( y_idx < ans.y_max && x_idx < ans.x_max ){ 
                store_method::__store<st_m>( ans[ block_z ][ y_idx ][ x_idx ] , sum );    
            }
        }

        template<int st_m>
        inline void conv2_full( GTensor3D &ans,
                                const GTensor3D &mat,
                                const GTensor4D &filter,
                                const GTensor1D &v_bias  ){
            if( filter.y_max <= Y_UNIT && filter.x_max <= MEM_UNIT ){
                int  grid_height= (ans.y_max+Y_UNIT-1  ) >> Y_UNIT_BITS ;
                int  grid_width = (ans.x_max+MEM_UNIT-1) >> MEM_UNIT_BITS;
                // pack 3D grid into 2D
                dim3 dimBlock( MEM_UNIT, Y_UNIT, 1 );
                dim3 dimGrid ( grid_width*grid_height, filter.h_max );
                
                __conv2_full_kernel_rec<st_m,Y_UNIT,MEM_UNIT> <<<dimGrid,dimBlock>>> ( grid_width, __GT(ans), __GT(mat), __GT(filter), __GT(v_bias) );
                
            }else{                
                error("too large filter size");
            }
        }        
    };

    // conv2_r_big_filter , restrict the filter size to be in (16,16)
    namespace cuda_tensor{
        /* calculate a block of convolution */  
        // restrict filter size to be in (Y_UNIT,MEM_UNIT)
        inline __device__ void __conv2_r_big_filter_block_procedure_rec_restricted( CUDA_CONV2_SUM_PARAM( sum, c_kahan ),
                                                                                    int   y_start, int x_start,
                                                                                    float s_ft [Y_UNIT][MEM_UNIT],
                                                                                    float s_mat[Y_UNIT<<1][MEM_UNIT<<1],
                                                                                    const __GT2D mat, 
                                                                                    const __GT2D filter ){
            // load in file
            __conv2::__load_mat_shared_pad_rec<Y_UNIT,MEM_UNIT,false>
                ( s_ft, filter, y_start, x_start );
            // load in matrix 
            __conv2::__load_mat_shared_pad_rec<Y_UNIT<<1,MEM_UNIT<<1,false>
                ( s_mat, mat  , y_start, x_start );
            
            __syncthreads();
                                                            
            for( int dy = 0 ; dy < Y_UNIT ; dy ++ ){
                for( int dx = 0 ; dx < MEM_UNIT ; dx ++ ){
                    /* s_ft get by broadcast, mat has no bank conflit  */                         
                    __conv2::add_sum( CUDA_CONV2_SUM_ARG( sum, c_kahan ), 
                                      s_ft[ dy ][ dx ] * s_mat[ threadIdx.y + dy ][ threadIdx.x + dx ] ); 
                }
            }        
        }

        template<int st_m>
        inline __device__ void __conv2_r_big_filter_procedure_rec_restricted( float s_ft [Y_UNIT][MEM_UNIT],
                                                                              float s_mat[Y_UNIT<<1][MEM_UNIT<<1],
                                                                              __GT2D ans,
                                                                              const __GT2D mat, 
                                                                              const __GT2D filter ){
            CUDA_CONV2_SUM_VAR_DEF( sum, c_kahan );

            for( int yy = 0 ; yy < filter.y_max ; yy += Y_UNIT )
                for( int xx = 0 ; xx < filter.x_max ; xx += MEM_UNIT ){
                    __conv2_r_big_filter_block_procedure_rec_restricted
                        ( CUDA_CONV2_SUM_ARG( sum, c_kahan ), yy, xx, s_ft, s_mat, mat, filter );   
                    __syncthreads();
                }
                                                            
            if( threadIdx.y < ans.y_max && threadIdx.x < ans.x_max ){                
                store_method::__store<st_m>( ans[ threadIdx.y ][ threadIdx.x ], sum );
            }            
        }
                
        template<int st_m>
        __global__ void __conv2_r_big_filter_kernel_rec_restricted( __GT4D ans,
                                                                     const __GT3D mat, 
                                                                     const __GT3D filter ){
            __shared__ float s_ft [ Y_UNIT ][ MEM_UNIT ];
            __shared__ float s_mat[ Y_UNIT<<1 ][ MEM_UNIT<<1 ];
            __conv2_r_big_filter_procedure_rec_restricted<st_m>
                ( s_ft, s_mat, ans[ blockIdx.y ][ blockIdx.x ], mat[ blockIdx.y ], filter[ blockIdx.x ] );
        }

        template<int st_m>
        inline void conv2_r_big_filter_origin( GTensor4D &ans,
                                               const GTensor3D &mat,
                                               const GTensor3D &filter ){
            if( ans.y_max <= Y_UNIT && ans.x_max <= MEM_UNIT ){
                dim3 dimBlock( MEM_UNIT,Y_UNIT, 1 );
                dim3 dimGrid ( ans.z_max, ans.h_max, 1  );
                
                __conv2_r_big_filter_kernel_rec_restricted <st_m><<<dimGrid,dimBlock,0,cuda_async::get_stream(ans,mat,filter)>>> 
                    ( __GT(ans), __GT(mat), __GT(filter) );
            }else{
                error("too large answer size");
            }
        }        
    };
};

/*---------------we want to add optimization in this part to convolution -----------*/
namespace apex_tensor{
    namespace cuda_tensor{
        namespace __conv2{
            template< int x_size, int amount,bool pad, bool chk_lower>
            inline __device__ void __load_line_shared( float m_shared[amount],
                                                       const __GT1D m_global,
                                                       int x_start ){
                for( int x = 0 ; x < amount ; x += x_size ){
                    const int xx = x       + threadIdx.x;
                    const int cx = x_start + xx;
                    if( cx < m_global.x_max && ((!chk_lower)||cx>=0 ) ){
                        m_shared[ xx ] = m_global[ cx ]; 
                    }else{
                        if( pad ) m_shared[ xx ] = 0.0f;
                    }
                }                
            }
            template< int x_size, int amount>
            inline __device__ void __fill_zero( float m_shared[amount] ){
                for( int x = 0 ; x < amount ; x += x_size ){
                    m_shared[ x+ threadIdx.x ] = 0.0f;
                }
            }
        };
    };

    namespace cuda_tensor{       
        template<int y_size,int x_bits>
        inline __device__ void __conv2_r_valid_opt_A( float &sum,
                                                      int y_start,
                                                      int x_start,
                                                      float s_ft[y_size       ][1<< x_bits],    
                                                      float s_mm[(y_size<<1)-1][1<<(x_bits+1)],
                                                      int ans_y_max, int ans_x_max,
                                                      const __GT2D mat,
                                                      const __GT2D filter ){            
            const int x_size = 1 << x_bits;
            const int yy = y_start + threadIdx.y;
            const int xx = x_start + threadIdx.x;
            __conv2::__load_line_shared<x_size,x_size,false,false> 
                ( s_ft[threadIdx.y], filter[threadIdx.y], 0 );
            
            if( yy < mat.y_max ){
                __conv2::__load_line_shared<x_size,x_size<<1,false,false>
                    ( s_mm[threadIdx.y], mat[yy], x_start );
            }            
            if( threadIdx.y != y_size-1 && yy + y_size < mat.y_max )
                __conv2::__load_line_shared<x_size,x_size<<1,false,false>
                    ( s_mm[threadIdx.y+y_size], mat[yy+y_size], x_start );
            
            __syncthreads();
            
            if( yy < ans_y_max && xx < ans_x_max ){
                for( int dy = 0; dy < y_size; dy ++ ){
                    float ss = 0.0f;
                    for( int dx = 0; dx < filter.x_max ; dx ++ )
                        ss += s_ft[dy][dx] * s_mm[ threadIdx.y + dy ][ threadIdx.x + dx ];
                    sum += ss;
                }
            }               
            __syncthreads();
        }
        
        template<int st_m,int y_size,int x_bits>
        __global__ void __conv2_r_valid_opt_A_kernel( int grid_width,
                                                      __GT3D ans,                                                   
                                                      const __GT3D mat,
                                                      const __GT4D filter,
                                                      const __GT1D h_bias   ){
            const int x_size = 1 << x_bits;
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;
            const int y_start = block_y *  y_size;
            const int x_start = block_x << x_bits;
            
            __shared__ float bias;
            __shared__ float s_ft[y_size   ][x_size];
            __shared__ float s_mm[(y_size<<1)-1][x_size<<1];

            //load the bias
            if( threadIdx.y == y_size-1 && threadIdx.x == x_size-1 ){
                // we use last thread because last thread seems more likely to be idle
                // no need to sync because sync will occur in latter procedure
                bias = h_bias[ block_z ];
            }

            float sum = 0.0f;
            for( int v = 0; v < mat.z_max ; v ++ )
                __conv2_r_valid_opt_A<y_size,x_bits>
                    ( sum, y_start, x_start, s_ft, s_mm, ans.y_max, ans.x_max, mat[v], filter[v][block_z] );

            const int yy = y_start + threadIdx.y;
            const int xx = x_start + threadIdx.x;
            
            if( yy < ans.y_max && xx < ans.x_max ){
                store_method::__store<st_m>( ans[block_z][yy][xx] , sum+bias );
            }
        }
        
        template<int st_m,int y_size>
        inline void conv2_r_valid_opt_A( GTensor3D &ans,
                                         const GTensor3D &mat,
                                         const GTensor4D &filter,
                                         const GTensor1D &h_bias ){
            const int x_bits = MEM_UNIT_BITS; 
            const int x_size = 1 << x_bits;
            const int y_max  = ( ans.y_max + y_size - 1 )/y_size;
            const int x_max  = ( ans.x_max + x_size - 1 )/x_size;
            
            dim3 dimBlock( x_size, y_size );
            dim3 dimGrid ( y_max*x_max , ans.z_max );

            __conv2_r_valid_opt_A_kernel<st_m,y_size,x_bits> <<<dimGrid,dimBlock>>>
                ( x_max, __GT(ans), __GT(mat), __GT(filter), __GT(h_bias) ); 
        }
                
        template<int st_m>
        inline void conv2_r_valid( GTensor3D &ans,
                                   const GTensor3D &mat,
                                   const GTensor4D &filter,
                                   const GTensor1D &h_bias ){
#if __CUDA_CONV2_USE_OPT__
            if( filter.x_max < MEM_UNIT ){ 
                switch( filter.y_max ){
                case 10: conv2_r_valid_opt_A<st_m,10>( ans, mat, filter, h_bias ); break;
                case 12: conv2_r_valid_opt_A<st_m,12>( ans, mat, filter, h_bias ); break;

                default: conv2_r_valid_orign<st_m>( ans, mat, filter, h_bias );    break;
                }
            }else{
                error("too large answer size");
            }
#else
            conv2_r_valid_orign<st_m>( ans, mat, filter, h_bias ); 
#endif
        }
    };
    
    /* conv2 r big filter */
    namespace cuda_tensor{
        template<int st_m,int y_size,int x_bits>
        inline __device__ void __conv2_r_big_filter_optA( float s_ft [y_size][(1<<x_bits)],
                                                          float s_mm [(y_size<<1)-1][1<<(x_bits+1)],
                                                          float s_rst[y_size][1<<x_bits],
                                                          __GT2D ans,
                                                          const __GT2D mat,
                                                          const __GT2D filter ){
            float sum = 0.0f;
            const int x_size = 1<<x_bits;

            for( int y_start = 0; y_start < filter.y_max; y_start += y_size )
                for( int x_start = 0; x_start < filter.x_max; x_start += (1<<x_bits) ){
                    // load filter data 
                    if( y_start+threadIdx.y < filter.y_max ){
                        __conv2::__load_line_shared<x_size,x_size,true,false> 
                            ( s_ft[threadIdx.y], filter[y_start+threadIdx.y], x_start );
                    }else{
                        __conv2::__fill_zero<x_size,x_size>( s_ft[ threadIdx.y ] );
                    }
                    if( y_start+threadIdx.y < mat.y_max ){
                        __conv2::__load_line_shared<x_size,x_size<<1,true,false> 
                            ( s_mm[threadIdx.y], mat[y_start+threadIdx.y], x_start );                        
                    }else{
                        __conv2::__fill_zero<x_size,x_size<<1 >( s_mm[ threadIdx.y] );
                    }
                    if( threadIdx.y != y_size-1 ){
                        if( y_start+threadIdx.y+y_size < mat.y_max ){
                            __conv2::__load_line_shared<x_size,x_size<<1,true,false> 
                                ( s_mm[threadIdx.y+y_size], mat[y_start+threadIdx.y+y_size], x_start );                        
                        }else{
                            __conv2::__fill_zero<x_size,x_size<<1 >( s_mm[ threadIdx.y + y_size ] );
                        }                   
                    }
                    __syncthreads();

                    // calculate multiplication
                    for( int x = 0; x < ans.x_max; x++ ){
                        float ss = 0.0f;
                        for( int y = 0; y < y_size; y ++ ){
                            // s_ft: no bank conflict, s_mm no bank conflict
                            ss += s_mm[ threadIdx.y+y ][ threadIdx.x+x ] * s_ft[ y ][ threadIdx.x ];
                        }                                  
                        s_rst[ threadIdx.y ][ threadIdx.x ] = ss;

                        // reduce sum 
                        __syncthreads();
                        cuda_reduce::reduce_1D<cuda_reduce::SUM,x_bits>( s_rst[threadIdx.y] );
                        __syncthreads();
                        if( threadIdx.x == x ){
                            sum += s_rst[ threadIdx.y ][ 0 ];
                        }               
                    }
                }
            if( threadIdx.x < ans.x_max ){
                store_method::__store<st_m>( ans[threadIdx.y][threadIdx.x], sum );
            }
        } 
                   
        // optimized using rolling array to cache data which reduces memory access 
        template<int st_m,int y_size,int x_bits>
        inline __device__ void __conv2_r_big_filter_optB( float s_ft [y_size][(1<<x_bits)],
                                                          float s_mm [(y_size<<1)-1][1<<(x_bits+1)],
                                                          float s_rst[y_size][1<<x_bits],
                                                          __GT2D ans,
                                                          const __GT2D mat,
                                                          const __GT2D filter ){
            float sum = 0.0f;
            const int x_size = 1<<x_bits;
            const int x_mask = (1<<(x_bits+1)) - 1;

            for( int y_start = 0; y_start < filter.y_max; y_start += y_size ){
                // load first part of data 
                if( y_start+threadIdx.y < mat.y_max ){
                    __conv2::__load_line_shared<x_size,x_size,true,false> 
                        ( s_mm[threadIdx.y], mat[y_start+threadIdx.y], 0 );                        
                }else{
                    __conv2::__fill_zero<x_size,x_size >( s_mm[ threadIdx.y] );
                }
                if( threadIdx.y != y_size-1 ){
                    if( y_start+threadIdx.y+y_size < mat.y_max ){
                        __conv2::__load_line_shared<x_size,x_size,true,false> 
                            ( s_mm[threadIdx.y+y_size], mat[y_start+threadIdx.y+y_size], 0 );                        
                    }else{
                        __conv2::__fill_zero<x_size,x_size >( s_mm[ threadIdx.y + y_size ] );
                    }                   
                }

                for( int x_start = 0; x_start < filter.x_max; x_start += x_size ){
                    // load filter data 
                    if( y_start+threadIdx.y < filter.y_max ){
                        __conv2::__load_line_shared<x_size,x_size,true,false> 
                            ( s_ft[threadIdx.y], filter[y_start+threadIdx.y], x_start );
                    }else{
                        __conv2::__fill_zero<x_size,x_size>( s_ft[ threadIdx.y ] );
                    }
                    // use rolling array
                    if( y_start+threadIdx.y < mat.y_max ){
                        __conv2::__load_line_shared<x_size,x_size,true,false> 
                            ( s_mm[threadIdx.y] + ((1-((x_start>>x_bits)&1)) << x_bits), 
                              mat[y_start+threadIdx.y], x_start + x_size );                        
                    }else{
                        __conv2::__fill_zero<x_size,x_size >( s_mm[ threadIdx.y ] + 
                                                              ((1-((x_start>>x_bits)&1)) << x_bits) );
                    }

                    if( threadIdx.y != y_size-1 ){
                        if( y_start+threadIdx.y+y_size < mat.y_max ){
                            __conv2::__load_line_shared<x_size,x_size,true,false> 
                                ( s_mm[threadIdx.y+y_size] + ((1-((x_start>>x_bits)&1)) << x_bits) , 
                                  mat[y_start+threadIdx.y+y_size], x_start + x_size );                        
                        }else{
                            __conv2::__fill_zero<x_size,x_size>( s_mm[ threadIdx.y + y_size ]+
                                                                     ((1-((x_start>>x_bits)&1)) << x_bits) );
                        }                   
                    }
                    __syncthreads();

                    // calculate multiplication
                    for( int x = 0; x < ans.x_max; x++ ){
                        float ss = 0.0f;
                        for( int y = 0; y < y_size; y ++ ){
                            // s_ft: no bank conflict, s_mm no bank conflict
                            ss += s_mm[ threadIdx.y+y ][ (threadIdx.x + x + (((x_start>>x_bits)&1)<<x_bits)) & x_mask  ] * s_ft[ y ][ threadIdx.x ];
                        }                                  
                        s_rst[ threadIdx.y ][ threadIdx.x ] = ss;

                        // reduce sum 
                        __syncthreads();
                        cuda_reduce::reduce_1D<cuda_reduce::SUM,x_bits>( s_rst[threadIdx.y] );
                        __syncthreads();
                        if( threadIdx.x == x ){
                            sum += s_rst[ threadIdx.y ][ 0 ];
                        }               
                    }
                }
            }
            if( threadIdx.x < ans.x_max ){
                store_method::__store<st_m>( ans[threadIdx.y][threadIdx.x], sum );
            }
        }                    
        
        template<int st_m,int y_size,int x_bits>
        __global__ void __conv2_r_big_filter_optA_kernel( __GT4D ans,
                                                          const __GT3D mat,
                                                          const __GT3D filter ){
            __shared__ float s_ft [y_size][(1<<x_bits)];
            __shared__ float s_mm [(y_size<<1)-1][1<<(x_bits+1)];
            __shared__ float s_rst[y_size][1<<x_bits];
            
            __conv2_r_big_filter_optB<st_m,y_size,x_bits>
                ( s_ft, s_mm, s_rst, ans[ blockIdx.y ][ blockIdx.x ], mat[ blockIdx.y ], filter[blockIdx.x] );  
        }
        

        template<int st_m,int y_size>
        inline void conv2_r_big_filter_optA( GTensor4D &ans,
                                             const GTensor3D &mat,
                                             const GTensor3D &filter ){
            const int x_bits = MEM_UNIT_BITS;
            const int x_size = 1 << x_bits;
            dim3 dimBlock( x_size, y_size, 1 );
            dim3 dimGrid ( ans.z_max, ans.h_max, 1  );
                            
            __conv2_r_big_filter_optA_kernel<st_m,y_size,x_bits><<<dimGrid,dimBlock,0,cuda_async::get_stream(ans,mat,filter)>>> 
                ( __GT(ans), __GT(mat), __GT(filter) );            
        }                        

        template<int st_m>
        inline void conv2_r_big_filter( GTensor4D &ans,
                                        const GTensor3D &mat,
                                        const GTensor3D &filter ){
#if __CUDA_CONV2_USE_OPT__
            if( ans.x_max < MEM_UNIT ){ 
                switch( ans.y_max ){
                case 10: conv2_r_big_filter_optA<st_m,10>( ans, mat, filter ); break;
                case 12: conv2_r_big_filter_optA<st_m,12>( ans, mat, filter ); break;
                default: conv2_r_big_filter_origin<st_m> ( ans, mat, filter ); break;
                }
            }else{
                error("too large answer size");
            }
#else
            conv2_r_big_filter_origin<st_m> ( ans, mat, filter );
#endif
        }        
    };
};

#endif
