#include "hip/hip_runtime.h"
#ifndef _CUDA_TENSOR_POOLING_CU_
#define _CUDA_TENSOR_POOLING_CU_

#include "cuda_tensor.cuh"
#include "base/cuda_reduce.cuh"

// pooling procedure that summary up information
namespace apex_tensor{
    namespace cuda_tensor{        
        /*----------the following are sum kernels that calculate sum of the data-------*/
        template<int st_m,int mapm_A, int mapm_B>
        __global__ void __tensor_sum_2D_kernel( float *v_reduce_A , float *v_reduce_B, const __GT3D src ){
            __shared__ float s_mmx[16][16];
            const __GT2D ss = src[ blockIdx.x ];
            
            float sumA = 0.0f, sumB = 0.0f;

            for( int yy = 0 ; yy < ss.y_max ; yy += 16 )
                for( int xx = 0 ; xx < ss.x_max ; xx += 16 ){
                    const int y_idx = yy + threadIdx.y; 
                    const int x_idx = xx + threadIdx.x; 

                    if( y_idx < ss.y_max && x_idx < ss.x_max ){
                        float a = ss[y_idx][x_idx];
                        sumA += map_method_A::__map<mapm_A>( a );
                        sumB += map_method_A::__map<mapm_B>( a );                    
                    }
                }
            
            s_mmx[ threadIdx.y ][ threadIdx.x ] = sumA;
            __syncthreads();
            
            cuda_reduce::reduce_2D<cuda_reduce::SUM,4,4>( s_mmx );
            // we only depend on thread 0 0, no need to sync

            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( v_reduce_A[ blockIdx.x ], s_mmx[0][0] ); 
            }
            
            s_mmx[ threadIdx.y ][ threadIdx.x ] = sumB;
            __syncthreads();
            
            cuda_reduce::reduce_2D<cuda_reduce::SUM,4,4>( s_mmx );
            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( v_reduce_B[ blockIdx.x ], s_mmx[0][0] ); 
            }
        }
               
        template<int st_m,int mapm_A, int mapm_B>
        inline void tensor_sum_2D( GTensor1D &ra, GTensor1D &rb, const GTensor3D &src ){
            dim3 dimBlock( 16, 16 );
            dim3 dimGrid ( src.z_max, 1 );
            __tensor_sum_2D_kernel<st_m,mapm_A,mapm_B> <<<dimGrid,dimBlock,0,cuda_async::get_stream(ra,rb,src)>>>
                ( ra.elem, rb.elem, __GT(src) );
        }
    };
    
    // mat sum 
    namespace cuda_tensor{
        // produce sum of matrix, store sum at s_mmx[0][0]
        template<int mapm>
        __device__ void __tensor_sum_2D_procedure( float s_mmx[16][16], const __GT2D ss ){
            float sum = 0.0f;
            for( int yy = 0; yy < ss.y_max; yy += 16 )
                for( int xx = 0; xx < ss.x_max; xx += 16 ){
                    const int y_idx = yy + threadIdx.y; 
                    const int x_idx = xx + threadIdx.x; 
                    
                    if( y_idx < ss.y_max && x_idx < ss.x_max ){
                        sum += map_method_A::__map<mapm>( ss[y_idx][x_idx] );
                    }
                }
            s_mmx[ threadIdx.y ][ threadIdx.x ] = sum;
            __syncthreads();            
            cuda_reduce::reduce_2D<cuda_reduce::SUM,4,4>( s_mmx );
        }
        
    
        template<int st_m, int map_m>
        __global__ void __tensor_sum_2D_kernel( float *v_reduce, const __GT3D src ){
            __shared__ float s_mmx[16][16];
            
            __tensor_sum_2D_procedure<map_m>( s_mmx, src[ blockIdx.x ] );
            // we only depend on thread 0 0
            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( v_reduce[ blockIdx.x ], s_mmx[0][0] ); 
            }
        }
        
        template<int st_m, int map_m>
        inline void tensor_sum_2D( GTensor1D &r , const GTensor3D &src ){
            dim3 dimBlock( 16 , 16 );
            dim3 dimGrid ( src.z_max ,1 );
            __tensor_sum_2D_kernel<st_m,map_m><<<dimGrid,dimBlock,0,cuda_async::get_stream(r,src)>>>( r.elem, __GT(src) );
        }

        template<int st_m, int map_m>
        __global__ void __tensor_sum_2D_kernel( __GT2D dst , const __GT4D src ){
            __shared__ float s_mmx[16][16];
            
            __tensor_sum_2D_procedure<map_m>( s_mmx, src[ blockIdx.y ][ blockIdx.x ] );
            // we only depend on thread 0 0
            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( dst[ blockIdx.y ][ blockIdx.x ], s_mmx[0][0] ); 
            }
        }
        
        template<int st_m, int map_m>
        inline void tensor_sum_2D( GTensor2D &r , const GTensor4D &src ){
            dim3 dimBlock( 16, 16 );
            dim3 dimGrid ( src.z_max, src.h_max );
            __tensor_sum_2D_kernel<st_m,map_m><<<dimGrid,dimBlock,0,cuda_async::get_stream(r,src)>>>( __GT(r), __GT(src) );
        }
    };

    // pool 
    namespace cuda_tensor{
        /* 
           pool src to dst,optimized for pool_size = 2^pool_bits, write the result to sum
           with block shape < 16 , 16 >
        */
        template<int pool_bits>
        __device__ void __pool_procedure_1616( float &sum,
                                               int block_y,
                                               int block_x,    
                                               float s_mm[16][17],
                                               const __GT2D src  ){
            // load from src 
            for( int y = 0 ; y < (1<<pool_bits) ; y ++ )
                for( int x = 0 ; x < (1<<pool_bits) ; x ++ ){                
                    int y_idx = block_y * (16 << pool_bits) + (y<<4) + threadIdx.y; 
                    int x_idx = block_x * (16 << pool_bits) + (x<<4) + threadIdx.x;

                    if( y_idx < src.y_max && x_idx < src.x_max ) {
                        s_mm[ threadIdx.y ][ threadIdx.x ] = src[ y_idx ][ x_idx ];
                    }else{
                        s_mm[ threadIdx.y ][ threadIdx.x ] = 0.0f; 
                    }
                    __syncthreads();
                    // reduce the sum
                    cuda_reduce::reduce_block_1616<cuda_reduce::SUM,pool_bits,pool_bits>( s_mm );
                    __syncthreads();
                    
                    // if the thread is in this range 
                    if( y == ((threadIdx.y<<pool_bits)>>4) && x == ((threadIdx.x<<pool_bits)>>4) ){
                        // no bank conflict in the same pool, since we only access bank in the same row 
                        sum = s_mm[ (threadIdx.y<<pool_bits) & 15 ][ (threadIdx.x<<pool_bits) & 15 ]; 
                    }
                    // must sync here !!
                    __syncthreads();
                }
        }
        
        /* pooling kernel, using 3DGrid */
        template<int st_m, int map_m, int pool_bits>
        __global__ void __pool_kernel_1616( int grid_width, __GT3D dst, const __GT3D src ){
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;

            __shared__ float s_mm[ 16 ][ 17 ];
            
            // pool procedure
            float sum = 0.0f;
            __pool_procedure_1616<pool_bits>( sum, block_y, block_x, s_mm, src[block_z] );        

            // store result back 
            const int yy_idx = (block_y << 4) + threadIdx.y;
            const int xx_idx = (block_x << 4) + threadIdx.x;
            
            if( yy_idx < dst.y_max && xx_idx < dst.x_max  ){  
                float val =  map_method_A::__map<map_m>( sum );
                store_method::__store<st_m>( dst[ block_z ][ yy_idx ][ xx_idx ], val );
            }        
        }

        /* pooling data up */
        template<int st_m,int map_m,int pool_bits>
        inline void __pool_up_1616( GTensor3D &dst, const GTensor3D &src ){        
            int  grid_height= (dst.y_max+15) >> 4 ;
            int  grid_width = (dst.x_max+15) >> 4;

            dim3 dimBlock( 16 , 16 );
            dim3 dimGrid( grid_width*grid_height, src.z_max );           
            __pool_kernel_1616<st_m,map_m,pool_bits><<<dimGrid,dimBlock>>>( grid_width, __GT(dst), __GT(src) );
        }

        /* ordinary pooling */
        template<int pool_size>
        inline __device__ float __sum_block_ord( int y_start, int x_start, float s_mm[pool_size][pool_size*16] ){
            float sum = 0.0f;
            for( int y = y_start ; y < y_start + pool_size ; y ++ )
                for( int x = x_start ; x < x_start + pool_size*16 ; x ++ ){
                    sum += s_mm[y][x];
                }
            return sum;
        }

        /* 
           pool src to dst,optimized for pool_size 
           with block shape < pool_size , pool_size*16 >
        */
        template<int pool_size>
        __device__ void __pool_procedure_ord( float &sum,
                                              int block_y,
                                              int block_x,    
                                              float s_mm[pool_size][pool_size*16],
                                              const __GT2D src  ){
            for( int y = 0 ; y < pool_size ; y ++ )
                for( int x = 0 ; x < pool_size; x ++ ){                
                    int y_idx = block_y * pool_size*pool_size    + y*pool_size    + threadIdx.y; 
                    int x_idx = block_x * pool_size*pool_size*16 + x*pool_size*16 + threadIdx.x;

                    if( y_idx < src.y_max && x_idx < src.x_max ) {
                        s_mm[ threadIdx.y ][ threadIdx.x ] = src[ y_idx ][ x_idx ];
                    }else{
                        s_mm[ threadIdx.y ][ threadIdx.x ] = 0.0f; 
                    }

                    __syncthreads();

                    // if the thread is in this range 
                    if( y == threadIdx.y && x == (threadIdx.x>>4) ){
                        sum = __sum_block_ord<pool_size>( 0, ( threadIdx.x & 15 ) * pool_size , s_mm );
                    }
                    // must sync here !!
                    __syncthreads();
                }
        }
        
        /* pooling kernel, using 3DGrid */
        template<int st_m, int map_m, int pool_size>
        __global__ void __pool_kernel_ord( int grid_width, __GT3D dst, const __GT3D src ){
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;

            __shared__ float s_mm[ pool_size ][ pool_size*16 ];
            
            // pool procedure
            float sum = 0.0f;
            __pool_procedure_ord<pool_size>( sum, block_y, block_x, s_mm, src[block_z] );        

            // store result back 
            const int yy_idx = block_y*pool_size    + threadIdx.y;
            const int xx_idx = block_x*pool_size*16 + threadIdx.x;
            
            if( yy_idx < dst.y_max && xx_idx < dst.x_max  ){  
                float val =  map_method_A::__map<map_m>( sum );
                store_method::__store<st_m>( dst[ block_z ][ yy_idx ][ xx_idx ], val );
            }        
        }
        
        
        template<int st_m,int map_m,int pool_size>
        inline void __pool_up_ord( GTensor3D &dst, const GTensor3D &src ){        
            int  grid_height= (dst.y_max+pool_size-1   ) / pool_size ;
            int  grid_width = (dst.x_max+pool_size*16-1) / (pool_size*16);

            dim3 dimBlock( pool_size*16 , pool_size );
            dim3 dimGrid( grid_width*grid_height, src.z_max );           
            __pool_kernel_ord<st_m,map_m,pool_size><<<dimGrid,dimBlock>>>( grid_width, __GT(dst), __GT(src) ); 
        }

        template<int st_m,int map_m>
        inline void pool_up( GTensor3D &dst, const GTensor3D &src, int pool_size ){
            switch( pool_size ){
            case 1 : map_A<st_m,map_m,GTensor3D> ( dst, src ); break;
            case 2 : __pool_up_1616<st_m,map_m,1>( dst, src ); break;
            case 3 : __pool_up_ord <st_m,map_m,3>( dst, src ); break;
            case 4 : __pool_up_1616<st_m,map_m,2>( dst, src ); break;
            case 5 : __pool_up_ord <st_m,map_m,5>( dst, src ); break;
            case 8 : __pool_up_1616<st_m,map_m,3>( dst, src ); break;                
            default: error("pool_size not supported"); 
            }
        }
    };

    // pooling sum 
    namespace cuda_tensor{
        /* 
           pool src, sum up the pooled value, then use two kind of maps to get mapped value, 
           store result to sumA, and sumB 
           ,optimized for pool_size = 2^pool_bits
           with block shape < 16 , 16 >       
        */
        template<int mapm_A, int mapm_B, int pool_bits, bool ceil_up>
        __device__ void __pool_sum_procedure_1616( float &sumA,         
                                                   float &sumB,
                                                   float s_mm[16][17],
                                                   const __GT2D src  ){
            int d_y_max, d_x_max;
            if( ceil_up ){
                d_y_max = (src.y_max + (1<<pool_bits)-1) >> pool_bits;
                d_x_max = (src.x_max + (1<<pool_bits)-1) >> pool_bits;

            }else{
                d_y_max = src.y_max >> pool_bits;
                d_x_max = src.x_max >> pool_bits;
            }
            
            for( int yy = 0 ; yy < d_y_max ; yy +=16 )
                for( int xx = 0 ; xx < d_x_max ; xx +=16 ){
                    float s = 0.0f;
                    // pool result 
                    __pool_procedure_1616<pool_bits>( s, yy>>4, xx>>4, s_mm, src );

                    const int yy_idx = yy + threadIdx.y;        
                    const int xx_idx = xx + threadIdx.x;

                    // add product 
                    if( yy_idx < d_y_max && xx_idx < d_x_max ) {
                        sumA += map_method_A::__map<mapm_A>( s ); 
                        sumB += map_method_A::__map<mapm_B>( s ); 
                    }
                } 
        }
        
        /* pooling kernel, using 3DGrid */
        template<int st_m, int mapm_A, int mapm_B, int pool_bits,bool ceil_up>
        __global__ void __pool_sum_kernel_1616( float *v_reduce_A, float *v_reduce_B, const __GT3D src ){
            __shared__ float s_mm[ 16 ][ 17 ];
                        
            float sumA = 0.0f;
            float sumB = 0.0f;  
            __pool_sum_procedure_1616<mapm_A,mapm_B,pool_bits, ceil_up>
                ( sumA, sumB, s_mm, src[blockIdx.x] );        
            
            float (*s_mmx)[16] = (float(*)[16])s_mm[0];
            
            s_mmx[ threadIdx.y ][ threadIdx.x ] = sumA;
            __syncthreads();
            
            cuda_reduce::reduce_2D<cuda_reduce::SUM,4,4>( s_mmx );
            // we only depend on thread 0 0
            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( v_reduce_A[ blockIdx.x ] , s_mmx[0][0] ); 
            }
            
            s_mmx[ threadIdx.y ][ threadIdx.x ] = sumB;
            __syncthreads();
            
            cuda_reduce::reduce_2D<cuda_reduce::SUM,4,4>( s_mmx );
            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( v_reduce_B[ blockIdx.x ] , s_mmx[0][0] ); 
            }
        }
        
        template<int st_m, int mapm_A, int mapm_B, int pool_bits, bool ceil_up>
        inline void __pool_sum_1616( GTensor1D &ra, GTensor1D &rb, const GTensor3D &src ){        
            dim3 dimBlock( 16 , 16 );
            dim3 dimGrid ( src.z_max ,1 );

            hipStream_t s = cuda_async::get_stream( ra, rb, src );
            __pool_sum_kernel_1616<st_m,mapm_A,mapm_B,pool_bits,ceil_up><<<dimGrid,dimBlock,0,s>>>( ra.elem, rb.elem, __GT(src) );
        }

        template<int mapm_A, int mapm_B, int pool_size, bool ceil_up>
        __device__ void __pool_sum_procedure_ord( float &sumA,         
                                                  float &sumB,
                                                  float s_mm[pool_size][pool_size*16],
                                                  const __GT2D src  ){
            int d_y_max, d_x_max;
            if( ceil_up ){
                d_y_max = (src.y_max + pool_size-1) / pool_size;
                d_x_max = (src.x_max + pool_size-1) / pool_size;

            }else{
                d_y_max = src.y_max / pool_size;
                d_x_max = src.x_max / pool_size;
            }
            
            for( int yy = 0 ; yy < d_y_max ; yy += pool_size )
                for( int xx = 0 ; xx < d_x_max ; xx += pool_size*16 ){
                    float s = 0.0f;
                    __pool_procedure_ord<pool_size>( s, yy/pool_size, xx/(pool_size*16), s_mm, src );

                    const int yy_idx = yy + threadIdx.y;        
                    const int xx_idx = xx + threadIdx.x;

                    // add product 
                    if( yy_idx < d_y_max && xx_idx < d_x_max ) {
                        sumA += map_method_A::__map<mapm_A>( s ); 
                        sumB += map_method_A::__map<mapm_B>( s ); 
                    }
                } 
        }
        
        template<int st_m, int mapm_A, int mapm_B, int pool_size, bool ceil_up>
        __global__ void __pool_sum_kernel_ord( float *v_reduce_A, float *v_reduce_B, const __GT3D src ){
            __shared__ float s_mm[ pool_size ][ pool_size*16 ];
                        
            float sumA = 0.0f;
            float sumB = 0.0f;  
            __pool_sum_procedure_ord<mapm_A,mapm_B,pool_size, ceil_up>
                ( sumA, sumB, s_mm, src[blockIdx.x] );        
                               
            s_mm[ threadIdx.y ][ threadIdx.x ] = sumA;
            __syncthreads();
            
            cuda_reduce::reduce_2D_non_align<cuda_reduce::SUM,pool_size,pool_size*16>( s_mm );
            // we only depend on thread 0 0
            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( v_reduce_A[ blockIdx.x ] , s_mm[0][0] ); 
            }
            
            s_mm[ threadIdx.y ][ threadIdx.x ] = sumB;
            __syncthreads();
            
            cuda_reduce::reduce_2D_non_align<cuda_reduce::SUM,pool_size,pool_size*16>( s_mm );

            if( threadIdx.y == 0 && threadIdx.x == 0 ){
                store_method::__store<st_m>( v_reduce_B[ blockIdx.x ] , s_mm[0][0] ); 
            }
        }        

        template<int st_m, int mapm_A, int mapm_B, int pool_size, bool ceil_up>
        inline void __pool_sum_ord( GTensor1D &ra, GTensor1D &rb, const GTensor3D &src ){        
            dim3 dimBlock( pool_size*16, pool_size );
            dim3 dimGrid ( src.z_max ,1 );

            hipStream_t s = cuda_async::get_stream( ra, rb, src );
            __pool_sum_kernel_ord<st_m,mapm_A,mapm_B,pool_size,ceil_up><<<dimGrid,dimBlock,pool_size,s>>>( ra.elem, rb.elem, __GT(src) ); 
        }
        
        template<int st_m, int mapm_A, int mapm_B, bool ceil_up>
        inline void pool_sum( GTensor1D &ra, GTensor1D &rb, const GTensor3D &src, int pool_size ){        
            switch( pool_size ){
            case 1: tensor_sum_2D  <st_m,mapm_A,mapm_B>( ra, rb, src ); break;
            case 2: __pool_sum_1616<st_m,mapm_A,mapm_B,1,ceil_up>( ra, rb, src ); break;
            case 3: __pool_sum_ord <st_m,mapm_A,mapm_B,3,ceil_up>( ra, rb, src ); break;
            case 4: __pool_sum_1616<st_m,mapm_A,mapm_B,2,ceil_up>( ra, rb, src ); break;
            case 5: __pool_sum_ord <st_m,mapm_A,mapm_B,5,ceil_up>( ra, rb, src ); break;
            case 8: __pool_sum_1616<st_m,mapm_A,mapm_B,3,ceil_up>( ra, rb, src ); break;
            default: error("pooling size not supported");
            }
        }
    };        

    // normalize by maxpooling softmax
    namespace cuda_tensor{                
        // normalize the data start from y_start,x_start by exp
        // return the normalization constant for 1
        template<int pool_size, int y_size, int x_size>
        inline __device__ float __norm_maxpooling_step1( int y_start, int x_start, float s_mm[y_size][x_size] ){
            // get the max value of the data
            float smax = s_mm[y_start][x_start];
            for( int y = y_start ; y < y_start + pool_size ; y ++ )
                for( int x = x_start ; x < x_start + pool_size ; x ++ ){
                    if( smax < s_mm[y][x] ) smax = s_mm[y][x];
                }
            // map to exp
            for( int y = y_start ; y < y_start + pool_size ; y ++ )
                for( int x = x_start ; x < x_start + pool_size ; x ++ ){
                    s_mm[ y ][ x ] = expf( s_mm[ y ][ x ] - smax ); 
                }
            return expf( - smax );
        }

        template<int pool_size, int y_size, int x_size>
        inline __device__ void __norm_maxpooling_step2( int y_start, int x_start, float s_mm[y_size][x_size], float nm ){
            // get the max value of the data
            float sum = nm;
            for( int y = y_start ; y < y_start + pool_size ; y ++ )
                for( int x = x_start ; x < x_start + pool_size ; x ++ ){
                    sum += s_mm[ y ][ x ];
                }
            // map to exp
            for( int y = y_start ; y < y_start + pool_size ; y ++ )
                for( int x = x_start ; x < x_start + pool_size ; x ++ ){
                    s_mm[ y ][ x ] /= sum;
                }
        }

        /* 
           normalize the data by maxpooling with pool_size = 2^pool_bits
           with block shape < 16 , 16 >
        */
        template<int st_m,int pool_bits>
        __device__ void __norm_maxpooling_procedure_1616( int block_y,
                                                          int block_x,    
                                                          float s_mm[16][16],
                                                          __GT2D dst,
                                                          const __GT2D energy ){
            // load from src 
            for( int y = 0 ; y < (1<<pool_bits) ; y ++ )
                for( int x = 0 ; x < (1<<pool_bits) ; x ++ ){                                
                    int y_idx = block_y * (16 << pool_bits) + (y<<4) + threadIdx.y;
                    int x_idx = block_x * (16 << pool_bits) + (x<<4) + threadIdx.x;                    
                    bool is_valid   = y_idx < energy.y_max && x_idx < energy.x_max;
                    bool is_inrange = (y == ((threadIdx.y<<pool_bits)>>4) && x == ((threadIdx.x<<pool_bits)>>4) );
                    
                    // we don't need to sync here since each thread always use the same position                     
                    if( is_valid ){
                        s_mm[ threadIdx.y ][ threadIdx.x ] = energy[ y_idx ][ x_idx ];
                    }else{
                        s_mm[ threadIdx.y ][ threadIdx.x ] = -1e20f; 
                    }
                    __syncthreads();
                    
                    float nm;
                    // if the thread is in this range 
                    if( is_inrange ){
                        // no bank conflict in the same pool, since we only access bank in the same row 
                        nm = __norm_maxpooling_step1<1<<pool_bits,16,16>( (threadIdx.y<<pool_bits) &15, 
                                                                          (threadIdx.x<<pool_bits) &15,
                                                                          s_mm );                                                 
                    }
                    __syncthreads();

                    if( !is_valid ) {
                        s_mm[ threadIdx.y ][ threadIdx.x ] = 0.0f;
                    }
                    __syncthreads();
                    
                    if( is_inrange ){
                        // no bank conflict in the same pool, since we only access bank in the same row 
                        __norm_maxpooling_step2<1<<pool_bits,16,16>( (threadIdx.y<<pool_bits) &15, 
                                                                     (threadIdx.x<<pool_bits) &15,
                                                                     s_mm, nm );                                                 
                    }
                    __syncthreads();
                    
                    if( is_valid ) {
                        float s = s_mm[ threadIdx.y ][ threadIdx.x ];
                        store_method::__store<st_m>( dst[y_idx][x_idx], s );
                    }
                    // no need to sync
                }
        }
        
        /* pooling kernel, using 3DGrid */
        template<int st_m, int pool_bits>
        __global__ void __norm_maxpooling_kernel_1616( int grid_width, 
                                                       __GT3D dst, 
                                                       const __GT3D energy ){
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;
            
            __shared__ float s_mm[ 16 ][ 16 ];
            
            __norm_maxpooling_procedure_1616<st_m,pool_bits>
                (  block_y, block_x, s_mm, dst[block_z], energy[block_z] );
        }
        
        /* pooling data up */
        template<int st_m, int pool_bits>
        inline void __norm_maxpooling_1616( GTensor3D &dst, const GTensor3D &energy ){                    
            dim3 dimBlock( 16 , 16 );       
            const int d_y_max = (energy.y_max + (1<<pool_bits)-1) >> pool_bits;  
            const int d_x_max = (energy.x_max + (1<<pool_bits)-1) >> pool_bits;

            int  grid_height= (d_y_max+15) >> 4 ;        
            int  grid_width = (d_x_max+15) >> 4;

            dim3 dimGrid( grid_width*grid_height, energy.z_max );
            
            __norm_maxpooling_kernel_1616<st_m,pool_bits><<<dimGrid,dimBlock>>>( grid_width, __GT(dst), __GT(energy) );
        }

        /* 
           normalize the data by maxpooling with pool_size
           with block shape < pool_size , 16*pool_size >
        */
        template<int st_m,int pool_size>
        __device__ void __norm_maxpooling_procedure_ord( int block_y,
                                                         int block_x,    
                                                         float s_mm[pool_size][pool_size*16],
                                                         __GT2D dst,
                                                         const __GT2D energy ){
            // load from src 
            for( int y = 0 ; y < pool_size ; y ++ )
                for( int x = 0 ; x < pool_size ; x ++ ){                                
                    int y_idx = block_y * pool_size*pool_size    + y*pool_size    + threadIdx.y;
                    int x_idx = block_x * pool_size*pool_size*16 + x*pool_size*16 + threadIdx.x;                    
                    bool is_valid   = y_idx < energy.y_max && x_idx < energy.x_max;
                    bool is_inrange = ( y==threadIdx.y && x == (threadIdx.x>>4) );
                    
                    // we don't need to sync here since each thread always use the same position                     
                    if( is_valid ){
                        s_mm[ threadIdx.y ][ threadIdx.x ] = energy[ y_idx ][ x_idx ];
                    }else{
                        s_mm[ threadIdx.y ][ threadIdx.x ] = -1e20f; 
                    }
                    __syncthreads();
                    
                    float nm;
                    // if the thread is in this range 
                    if( is_inrange ){
                        nm = __norm_maxpooling_step1<pool_size,pool_size,pool_size*16>
                            ( 0, (threadIdx.x&15)*pool_size, s_mm );                                                 
                    }
                    __syncthreads();

                    if( !is_valid ) {
                        s_mm[ threadIdx.y ][ threadIdx.x ] = 0.0f;
                    }
                    __syncthreads();
                    
                    if( is_inrange ){
                        __norm_maxpooling_step2<pool_size,pool_size,pool_size*16>
                            ( 0, (threadIdx.x&15)*pool_size, s_mm, nm );                                                 
                    }
                    __syncthreads();
                    
                    if( is_valid ) {
                        float s = s_mm[ threadIdx.y ][ threadIdx.x ];
                        store_method::__store<st_m>( dst[y_idx][x_idx], s );
                    }
                }
        }

        template<int st_m, int pool_size>
        __global__ void __norm_maxpooling_kernel_ord( int grid_width, 
                                                      __GT3D dst, 
                                                      const __GT3D energy ){
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;
            
            __shared__ float s_mm[ pool_size ][ pool_size*16 ];
            
            __norm_maxpooling_procedure_ord<st_m,pool_size>
                (  block_y, block_x, s_mm, dst[block_z], energy[block_z] );
        }
        
        template<int st_m, int pool_size>
        inline void __norm_maxpooling_ord( GTensor3D &dst, const GTensor3D &energy ){                    
            dim3 dimBlock( pool_size*16, pool_size );       
            const int d_y_max = (energy.y_max + pool_size-1) / pool_size;  
            const int d_x_max = (energy.x_max + pool_size-1) / pool_size;

            int  grid_height= (d_y_max+ pool_size-1)    / pool_size;        
            int  grid_width = (d_x_max+ pool_size*16-1) / (pool_size*16);

            dim3 dimGrid( grid_width*grid_height, energy.z_max );
            
            __norm_maxpooling_kernel_ord<st_m,pool_size><<<dimGrid,dimBlock>>>( grid_width, __GT(dst), __GT(energy) );
        }
        
        template<int st_m>
        inline void norm_maxpooling( GTensor3D &dst, const GTensor3D &energy, int pool_size ){        
            switch( pool_size ){
            case 1: map_A<st_m,map_method_A::SIGMOID>( dst, energy ); break;
            case 2: __norm_maxpooling_1616<st_m,1>( dst, energy );    break;
            case 3: __norm_maxpooling_ord <st_m,3>( dst, energy );    break;
            case 4: __norm_maxpooling_1616<st_m,2>( dst, energy );    break;
            case 5: __norm_maxpooling_ord <st_m,5>( dst, energy );    break;
            case 8: __norm_maxpooling_1616<st_m,3>( dst, energy );    break;
            default: error("pooling size not supported");
            }
        }        
    };
};
#endif

