#include "hip/hip_runtime.h"
#ifndef _CUDA_TENSOR_SAMPLING_CU_
#define _CUDA_TENSOR_SAMPLING_CU_

#include "cuda_tensor.cuh"
#include "rand/cuda_rand.cuh"
#include "rand/cuda_sampling.cuh"

namespace apex_tensor{
    namespace cuda_tensor{        
        // sample binary using prob
        template<int st_m,int block_dim_bits>
        __global__ void sample_binary_kernel( float *elem_dst , const float *elem_src, 
                                              unsigned int pitch_dst, unsigned int pitch_src,
                                              int y_max       , int x_max,
                                              const float *rnd  ){
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;            
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst = get_line      ( elem_dst, y, pitch_dst );
            elem_src = get_line_const( elem_src, y, pitch_src );
                        
            if( y < y_max  && x < x_max ){
                float val = cuda_rand::sample_binary( elem_src[x], cuda_rand::get_rand( rnd, tid ) - 1.0f );
                store_method::__store<st_m>( elem_dst[x], val );
            }            
        }

        template<int st_m,typename T>
        inline void sample_binary( T &dst, const T &src ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            const float *rnd = cuda_rand::rand_singles( dimGrid.x * dimBlock.x ); 
            
            sample_binary_kernel<st_m,BASE_THREAD_BITS> <<<dimGrid,dimBlock>>>
                ( dst.elem, src.elem, dst.pitch,  src.pitch, y_max, x_max, rnd );
        } 

        template<int st_m,int block_dim_bits>
        __global__ void sample_recified_linear_kernel( float *elem_dst , const float *elem_src, 
                                                unsigned int pitch_dst, unsigned int pitch_src,
                                                int y_max       , int x_max,
                                                const float *rnd ){
            __shared__ float s_rnd[ 1<<block_dim_bits ];
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;            
            const float r = cuda_rand::sample_gaussian<block_dim_bits>( cuda_rand::get_rand(rnd,tid), threadIdx.x, s_rnd );
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst = get_line      ( elem_dst, y, pitch_dst );
            elem_src = get_line_const( elem_src, y, pitch_src );
                        
            if( y < y_max  && x < x_max ){
                float ans = elem_src[x] + r / ( 1.0f + expf( -elem_src[x] ) );
                if( ans < 0.0f ) ans = 0.0f;
                store_method::__store<st_m>( elem_dst[x], ans );
            }            
        }
        
        template<int st_m,typename T>
        inline void sample_recified_linear( T &dst, const T &src ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            const float *rnd = cuda_rand::rand_singles( dimGrid.x * dimBlock.x ); 
            
            sample_recified_linear_kernel<st_m,BASE_THREAD_BITS> <<<dimGrid,dimBlock>>>
                ( dst.elem, src.elem, dst.pitch,  src.pitch, y_max, x_max, rnd );
        } 

        // sample gaussian with given mean and sd
        template<int st_m,int block_dim_bits>
        __global__ void sample_gaussian_kernel( float *elem_dst , const float *elem_src, 
                                                unsigned int pitch_dst, unsigned int pitch_src,
                                                int y_max       , int x_max,
                                                const float *rnd, float sd ){
            __shared__ float s_rnd[ 1<<block_dim_bits ];
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;            
            const float r = cuda_rand::sample_gaussian<block_dim_bits>( cuda_rand::get_rand(rnd,tid), threadIdx.x, s_rnd ) * sd;
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst = get_line      ( elem_dst, y, pitch_dst );
            elem_src = get_line_const( elem_src, y, pitch_src );
                        
            if( y < y_max  && x < x_max ){
                store_method::__store<st_m>( elem_dst[x], elem_src[x] + r );
            }            
        }
        
        template<int st_m,typename T>
        inline void sample_gaussian( T &dst, const T &src, float sd ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            const float *rnd = cuda_rand::rand_singles( dimGrid.x * dimBlock.x ); 
            
            sample_gaussian_kernel<st_m,BASE_THREAD_BITS> <<<dimGrid,dimBlock>>>
                ( dst.elem, src.elem, dst.pitch,  src.pitch, y_max, x_max, rnd, sd );
        } 
        
        // sample gaussian
        template<int st_m,int block_dim_bits>
        __global__ void sample_gaussian_kernel( float *elem_dst ,
                                                unsigned int pitch_dst,
                                                int y_max       , int x_max,
                                                const float *rnd, float sd  ){
            __shared__ float s_rnd[ 1<<block_dim_bits ];
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;            
            const float r = cuda_rand::sample_gaussian<block_dim_bits>( cuda_rand::get_rand(rnd,tid), threadIdx.x, s_rnd ) * sd;            
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst = get_line( elem_dst, y, pitch_dst );
                        
            if( y < y_max  && x < x_max ){
                store_method::__store<st_m>( elem_dst[x], r );
            }            
        }
        
        template<int st_m,typename T>
        inline void sample_gaussian( T &dst, float sd ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            const float *rnd = cuda_rand::rand_singles( dimGrid.x * dimBlock.x ); 
            
            sample_gaussian_kernel<st_m,BASE_THREAD_BITS> <<<dimGrid,dimBlock>>>
                ( dst.elem, dst.pitch, y_max, x_max, rnd, sd );
        } 
                
        /* 
           sample maxpooling with pool_size = 2^pool_bits
           with block shape < Y_UNIT , X_UNIT >
        */
        template<int st_m,int pool_bits>
        __device__ void __sample_maxpooling_procedure_rec( int block_y,
                                                           int block_x,    
                                                           float s_mm[Y_UNIT][MEM_UNIT],
                                                           __GT2D dst,
                                                           const __GT2D prob,
                                                           const float *rnd ){
            float r = cuda_rand::get_rand( rnd, (threadIdx.y<<MEM_UNIT_BITS) + threadIdx.x ) - 1.0f;
            
            // load from src 
            for( int y = 0 ; y < (1<<pool_bits) ; y ++ )
                for( int x = 0 ; x < (1<<pool_bits) ; x ++ ){                                
                    int y_idx = block_y * (Y_UNIT   << pool_bits) + (y<<Y_UNIT_BITS)   + threadIdx.y;
                    int x_idx = block_x * (MEM_UNIT << pool_bits) + (x<<MEM_UNIT_BITS) + threadIdx.x;
                    
                    // we don't need to sync here since each thread always use the same position 
                    //__syncthreads();
                    
                    // load data into memory 
                    if( y_idx < prob.y_max && x_idx < prob.x_max ) {
                        s_mm[ threadIdx.y ][ threadIdx.x ] = prob[ y_idx ][ x_idx ];
                    }else{
                        s_mm[ threadIdx.y ][ threadIdx.x ] = 0.0f; 
                    }
                    __syncthreads();
                    
                    // if the thread is in this range 
                    if( y == ((threadIdx.y<<pool_bits)>>Y_UNIT_BITS) && x == ((threadIdx.x<<pool_bits)>>MEM_UNIT_BITS) ){
                        // no bank conflict in the same pool, since we only access bank in the same row 
                        cuda_rand::sample_maxpooling<pool_bits,MEM_UNIT>( (threadIdx.y<<pool_bits) &Y_UNIT_MASK, 
                                                                          (threadIdx.x<<pool_bits) &MEM_UNIT_MASK,
                                                                          s_mm, r );                                                 
                    }
                    __syncthreads();
                    
                    if( y_idx < dst.y_max && x_idx < dst.x_max ) {
                        float s = s_mm[ threadIdx.y ][ threadIdx.x ];
                        store_method::__store<st_m>( dst[y_idx][x_idx], s );
                    }
                }
        }
        
        /* pooling kernel, using 3DGrid */
        template<int st_m, int pool_bits>
        __global__ void __sample_maxpooling_rec_kernel_3DGrid( int grid_width, 
                                                               __GT3D dst, 
                                                               const __GT3D prob, 
                                                               const float *rnd ){
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;
            
            __shared__ float s_mm[ Y_UNIT ][ MEM_UNIT ];
            
            __sample_maxpooling_procedure_rec<st_m,pool_bits>
                (  block_y, block_x, s_mm, dst[block_z], prob[block_z], rnd + 
                   block_z*(gridDim.x<<(MEM_UNIT_BITS+Y_UNIT_BITS)) + (blockIdx.x<<(MEM_UNIT_BITS+Y_UNIT_BITS)) );        
        }
        
        template<int st_m, int pool_bits>
        inline void __sample_maxpooling_rec( GTensor3D &dst, const GTensor3D &prob ){
            dim3 dimBlock( MEM_UNIT , Y_UNIT );       
            const int d_y_max = (prob.y_max + (1<<pool_bits) - 1) >> pool_bits;  
            const int d_x_max = (prob.x_max + (1<<pool_bits) - 1) >> pool_bits;

            int  grid_height= (d_y_max+Y_UNIT-1  ) >> Y_UNIT_BITS ;        
            int  grid_width = (d_x_max+MEM_UNIT-1) >> MEM_UNIT_BITS;

            dim3 dimGrid( grid_width*grid_height, prob.z_max );
        
            const float *rnd  = cuda_rand::rand_singles( (dimGrid.y*dimGrid.x)<<(MEM_UNIT_BITS+Y_UNIT_BITS) );
            
            __sample_maxpooling_rec_kernel_3DGrid<st_m,pool_bits><<<dimGrid,dimBlock>>>( grid_width, __GT(dst), __GT(prob), rnd );
        }        
                
        /* 
           sample maxpooling with pool_size 
           with block shape < pool_size , 16*pool_size >
        */
        template<int st_m,int pool_size>
        __device__ void __sample_maxpooling_procedure_ord( int block_y,
                                                           int block_x,    
                                                           float s_mm[pool_size][MEM_UNIT*pool_size],
                                                           __GT2D dst,
                                                           const __GT2D prob,
                                                           const float *rnd ){
            float r = cuda_rand::get_rand( rnd, (threadIdx.y*pool_size*MEM_UNIT) + threadIdx.x ) - 1.0f;
            
            // load from src 
            for( int y = 0 ; y < pool_size ; y ++ )
                for( int x = 0 ; x < pool_size ; x ++ ){                                
                    int y_idx = block_y*pool_size*pool_size    + y*pool_size    + threadIdx.y;
                    int x_idx = block_x*pool_size*pool_size*MEM_UNIT + x*pool_size*MEM_UNIT + threadIdx.x;
                    
                    // we don't need to sync here since each thread always use the same position 
                    //__syncthreads();
                    
                    // load data into memory 
                    if( y_idx < prob.y_max && x_idx < prob.x_max ) {
                        s_mm[ threadIdx.y ][ threadIdx.x ] = prob[ y_idx ][ x_idx ];
                    }else{
                        s_mm[ threadIdx.y ][ threadIdx.x ] = 0.0f; 
                    }
                    __syncthreads();
                    
                    // if the thread is in this range 
                    if( y == threadIdx.y && x == (threadIdx.x>>MEM_UNIT_BITS) ){
                        // no bank conflict in the same pool, since we only access bank in the same row 
                        cuda_rand::sample_maxpooling_ord<pool_size,MEM_UNIT>( 0, 
                                                                              (threadIdx.x & MEM_UNIT_MASK) * pool_size,
                                                                              s_mm, r );                                                 
                    }
                    __syncthreads();
                    
                    if( y_idx < dst.y_max && x_idx < dst.x_max ) {
                        float s = s_mm[ threadIdx.y ][ threadIdx.x ];
                        store_method::__store<st_m>( dst[y_idx][x_idx], s );
                    }
                }
        }
        
        template<int st_m, int pool_size>
        __global__ void __sample_maxpooling_ord_kernel_3DGrid( int grid_width, 
                                                               __GT3D dst, 
                                                               const __GT3D prob, 
                                                               const float *rnd ){
            const int block_z = blockIdx.y;
            const int block_y = blockIdx.x / grid_width;
            const int block_x = blockIdx.x % grid_width;
            
            __shared__ float s_mm[ pool_size ][ pool_size*MEM_UNIT ];
            
            __sample_maxpooling_procedure_ord<st_m,pool_size>
                (  block_y, block_x, s_mm, dst[block_z], prob[block_z], 
                   rnd + block_z*(gridDim.x*pool_size*pool_size*MEM_UNIT) + (blockIdx.x*pool_size*pool_size*MEM_UNIT) );        
        }
        
        template<int st_m, int pool_size>
        inline void __sample_maxpooling_ord( GTensor3D &dst, const GTensor3D &prob ){
            dim3 dimBlock( pool_size*MEM_UNIT, pool_size );       

            const int d_y_max = (prob.y_max + pool_size-1) / pool_size;  
            const int d_x_max = (prob.x_max + pool_size-1) / pool_size;

            int  grid_height= (d_y_max+pool_size   -1) / pool_size;        
            int  grid_width = (d_x_max+pool_size*MEM_UNIT-1) / (pool_size*MEM_UNIT);

            dim3 dimGrid( grid_width*grid_height, prob.z_max );
        
            const float *rnd  = cuda_rand::rand_singles( (dimGrid.y*dimGrid.x)*(pool_size*pool_size*MEM_UNIT) );
            
            __sample_maxpooling_ord_kernel_3DGrid<st_m,pool_size><<<dimGrid,dimBlock>>>( grid_width, __GT(dst), __GT(prob), rnd );
        }        
        
        /* pooling data up */
        template<int st_m>
        inline void sample_maxpooling( GTensor3D &dst, const GTensor3D &prob, int pool_size ){        
            switch( pool_size ){
            case 1: sample_binary<st_m>( dst, prob );              break;
            case 2: __sample_maxpooling_rec<st_m,1>( dst, prob ); break;   
            case 3: __sample_maxpooling_ord <st_m,3>( dst, prob ); break;   
            case 4: __sample_maxpooling_rec<st_m,2>( dst, prob ); break;   
            case 8: __sample_maxpooling_rec<st_m,3>( dst, prob ); break;   
            default: error("pooling size not supported");
            }
        }                        
    };
};
#endif

