#include "hip/hip_runtime.h"
#ifndef _CUDA_TENSOR_OP_CU_
#define _CUDA_TENSOR_OP_CU_

#include "cuda_tensor.cuh"
namespace apex_tensor{
    namespace cuda_tensor{
        template<int st_m,int block_dim_bits>
        __global__ void store_kernel( float *elem , 
                                      size_t pitch,
                                      int    y_max, int x_max,
                                      float  src ){
            const int tid     = (blockIdx.x << block_dim_bits) + threadIdx.x;            
            const int x_mm    = get_align_width( x_max );
            const int y       = tid / x_mm;
            const int x       = tid % x_mm;

            elem = get_line( elem, y, pitch );  
                        
            if( y < y_max  && x < x_max ){
                store_method::__store<st_m>( elem[ x ] , src );
            }            
        }

        // store one element to another 
        template<int st_m,typename T>
        inline void store( T &ts, float src ){
            int stride = get_align_width( ts.x_max );
            int y_max  = num_line( ts );
            int x_max  = ts.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            
            store_kernel<st_m,BASE_THREAD_BITS> <<<dimGrid,dimBlock,0,cuda_async::get_stream(ts)>>>
                ( ts.elem, ts.pitch, y_max, x_max, src );
        }  
        
        // test pass
        template<int st_m,int mapm_A,int block_dim_bits>
        __global__ void map_A_kernel( float *elem_dst , const float *elem_src, 
                                      size_t pitch_dst, size_t pitch_src,
                                      int y_max       , int x_max ){
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst = get_line      ( elem_dst, y, pitch_dst );
            elem_src = get_line_const( elem_src, y, pitch_src );
            
            
            if( y < y_max && x < x_max ){
                float val = map_method_A::__map<mapm_A>( elem_src[x] );
                store_method::__store<st_m>( elem_dst[x], val );
            }            
        }
        
        template<int st_m,int mapm_A,typename T>
        inline void map_A( T &dst, const T &src ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            
            map_A_kernel<st_m,mapm_A,BASE_THREAD_BITS> <<<dimGrid,dimBlock,0,cuda_async::get_stream(dst,src)>>>
                ( dst.elem, src.elem, dst.pitch,  src.pitch, y_max, x_max );
        } 
                
        // takes a source and a float 
        template<int st_m,int mapm_B,int block_dim_bits>
        __global__ void map_B_kernel( float *elem_dst , const float *elem_src, 
                                      size_t pitch_dst, size_t pitch_src,
                                      int y_max       , int x_max, float src_b ){
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst = get_line      ( elem_dst, y, pitch_dst );
            elem_src = get_line_const( elem_src, y, pitch_src );
            
            
            if( y < y_max  && x < x_max ){
                float val = map_method_B::__map<mapm_B>( elem_src[x], src_b );
                store_method::__store<st_m>( elem_dst[x] , val );
            }            
        }
        
        template<int st_m,int mapm_B,typename T>
        inline void map_B( T &dst, const T &src, float src_b ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            
            map_B_kernel<st_m,mapm_B,BASE_THREAD_BITS> <<<dimGrid,dimBlock,0,cuda_async::get_stream(dst,src)>>>
                ( dst.elem, src.elem, dst.pitch,  src.pitch, y_max, x_max, src_b );
        }          

        // takes two source and a float 
        template<int st_m,int mapm_B,int block_dim_bits>
        __global__ void map_C_kernel( float *elem_dst , 
                                      const float *elem_srca, const float *elem_srcb, 
                                      size_t pitch_dst, size_t pitch_srca, size_t pitch_srcb,
                                      int y_max       , int x_max ){
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst  = get_line       ( elem_dst, y, pitch_dst );
            elem_srca = get_line_const( elem_srca, y, pitch_srca );
            elem_srcb = get_line_const( elem_srcb, y, pitch_srcb );
            
            
            if( y < y_max  && x < x_max ){
                float val = map_method_B::__map<mapm_B>( elem_srca[x], elem_srcb[x] );
                store_method::__store<st_m>( elem_dst[x] , val );
            }            
        }
        
        template<int st_m,int mapm_B,typename T>
        inline void map_C( T &dst, const T &srca, const T &srcb ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            
            map_C_kernel<st_m,mapm_B,BASE_THREAD_BITS> <<<dimGrid,dimBlock,0,cuda_async::get_stream(dst,srca,srcb)>>>
                ( dst.elem, srca.elem, srcb.elem, dst.pitch, srca.pitch, srcb.pitch, y_max, x_max );
        }          

        // takes two source and a float 
        template<int st_m,int mapm_D,int block_dim_bits>
        __global__ void map_D_kernel( float *elem_dst , 
                                      const float *elem_srca, const float *elem_srcb, 
                                      size_t pitch_dst, size_t pitch_srca, size_t pitch_srcb,
                                      int y_max       , int x_max,
                                      float sa        , float sb   ){
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;
            const int x_mm= get_align_width( x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;
            
            elem_dst  = get_line       ( elem_dst, y, pitch_dst );
            elem_srca = get_line_const( elem_srca, y, pitch_srca );
            elem_srcb = get_line_const( elem_srcb, y, pitch_srcb );
            
            
            if( y < y_max  && x < x_max ){
                float val = map_method_D::__map<mapm_D>( elem_srca[x], elem_srcb[x], sa, sb );
                store_method::__store<st_m>( elem_dst[x] , val );
            }            
        }
        
        template<int st_m,int mapm_D,typename T>
        inline void map_D( T &dst, const T &srca, const T &srcb, float sa, float sb ){
            int stride     = get_align_width( dst.x_max );
            int y_max      = num_line( dst );
            int x_max      = dst.x_max;
            
            int num_block = (y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , 1, 1 );
            
            map_D_kernel<st_m,mapm_D,BASE_THREAD_BITS> <<<dimGrid,dimBlock,0,cuda_async::get_stream(dst,srca,srcb)>>>
                ( dst.elem, srca.elem, srcb.elem, dst.pitch, srca.pitch, srcb.pitch, y_max, x_max, sa, sb );
        }                  

        
        template<int st_m,int mapm_B,int block_dim_bits >
        __global__ void map_E_kernel( __GT4D dst,
                                      const __GT2D src, float src_b ){
            const int block_y = blockIdx.y / dst.z_max;
            const int block_x = blockIdx.y % dst.z_max;
        
            __shared__ float src_a;
            if( threadIdx.x == 0 ){
                src_a = src[block_y][block_x];
            }
            __GT2D dd = dst[block_y][block_x];
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;
            const int x_mm= get_align_width( dd.x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;                                                
            
            __syncthreads();

            if( y < dd.y_max  && x < dd.x_max ){
                float val = map_method_B::__map<mapm_B>( src_a, src_b );
                store_method::__store<st_m>( dd[y][x] , val );
            }            
        }
        
        template<int st_m,int mapm_B>
        inline void map_E( GTensor4D &dst, const GTensor2D &src, float src_b ){
            int stride    = get_align_width( dst.x_max );            
            int num_block = (dst.y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , dst.z_max*dst.h_max, 1 );
            
            map_E_kernel<st_m,mapm_B,BASE_THREAD_BITS> <<<dimGrid,dimBlock,0,cuda_async::get_stream(dst,src)>>>
                ( __GT(dst), __GT(src), src_b );
        }          
        
        
        // map the edge area only
        template<int st_m,int mapm_A,int block_dim_bits>
        __global__ void map_A_edge_kernel( __GT3D dst , const __GT3D src, 
                                           int edge_y_len, int edge_x_len ){
            __GT2D dd = dst[ blockIdx.y ];
            const __GT2D ss = src[ blockIdx.y ];
            const int tid = (blockIdx.x << block_dim_bits) + threadIdx.x;
            const int x_mm= get_align_width( dd.x_max );
            const int y   = tid / x_mm;
            const int x   = tid % x_mm;

            if( y < dd.y_max && x < dd.x_max )
                if( y < edge_y_len || y >= dd.y_max - edge_y_len ||
                    x < edge_x_len || x >= dd.x_max - edge_x_len  ){
                        float val = map_method_A::__map<mapm_A>( ss[y][x] );
                        store_method::__store<st_m>( dd[y][x], val );
                }            
        }
        
        template<int st_m,int mapm_A>
        inline void map_A_edge( GTensor3D &dst, const GTensor3D &src, int edge_y_len, int edge_x_len ){
            int stride     = get_align_width( dst.x_max );            
            int num_block = (dst.y_max*stride + BASE_THREAD_NUM-1)/BASE_THREAD_NUM;

            dim3 dimBlock( BASE_THREAD_NUM, 1, 1 );
            dim3 dimGrid ( num_block      , dst.z_max , 1 );
            
            map_A_edge_kernel<st_m,mapm_A,BASE_THREAD_BITS> <<<dimGrid,dimBlock,0,cuda_async::get_stream(dst,src)>>>
                ( __GT(dst), __GT(src), edge_y_len, edge_x_len );
        } 
    };
};

#endif
