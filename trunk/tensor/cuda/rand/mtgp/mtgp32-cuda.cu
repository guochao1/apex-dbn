#include "hip/hip_runtime.h"
/**
 * @file mtgp32-cuda.cu
 *
 * @brief Sample Program for CUDA 2.2
 *
 * MTGP32-23209
 * This program generates 32-bit unsigned integers.
 * The period of generated integers is 2<sup>23209</sup>-1.
 *
 * This also generates single precision floating point numbers
 * uniformly distributed in the range [1, 2). (float r; 1.0 <= r < 2.0)
 *
 * @author Mutsuo Saito (Hiroshima University)
 * @author Makoto Matsumoto (Hiroshima University)
 *
 * Copyright (C) 2009 Mutsuo Saito, Makoto Matsumoto and
 * Hiroshima University. All rights reserved.
 *
 * The new BSD License is applied to this software, see LICENSE.txt
 */
#define __STDC_FORMAT_MACROS 1
#define __STDC_CONSTANT_MACROS 1
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <cutil.h>

#ifdef _MSC_VER
#include "stdint.h"
#include "inttypes.h"
#else
#include <stdint.h>
#include <inttypes.h>
#endif


extern "C" {
#include "mtgp32-fast.h"
}
#define MEXP 23209
#define N 726
#define THREAD_NUM 512
#define LARGE_SIZE (THREAD_NUM * 3)
#define BLOCK_NUM 32	     /* You can change this value up to 128 */
#define TBL_SIZE 16


/**
 * kernel I/O
 * This structure must be initialized before first use.
 */
struct mtgp32_kernel_status_t {
    uint32_t status[N];
};

/*
 * Generator Parameters.
 */
__constant__ uint32_t param_tbl[BLOCK_NUM][TBL_SIZE];
__constant__ uint32_t temper_tbl[BLOCK_NUM][TBL_SIZE];
__constant__ uint32_t single_temper_tbl[BLOCK_NUM][TBL_SIZE];
__constant__ uint32_t pos_tbl[BLOCK_NUM];
__constant__ uint32_t sh1_tbl[BLOCK_NUM];
__constant__ uint32_t sh2_tbl[BLOCK_NUM];
/* high_mask and low_mask should be set by make_constant(), but
 * did not work.
 */
__constant__ uint32_t mask = 0xff800000;

/**
 * Shared memory
 * The generator's internal status vector.
 */
__shared__ uint32_t status[LARGE_SIZE]; /* 512 * 3 elements, 6144 bytes. */

/**
 * The function of the recursion formula calculation.
 *
 * @param[in] X1 the farthest part of state array.
 * @param[in] X2 the second farthest part of state array.
 * @param[in] Y a part of state array.
 * @param[in] bid block id.
 * @return output
 */
__device__ uint32_t para_rec(uint32_t X1, uint32_t X2, uint32_t Y, int bid) {
    uint32_t X = (X1 & mask) ^ X2;
    uint32_t MAT;

    X ^= X << sh1_tbl[bid];
    Y = X ^ (Y >> sh2_tbl[bid]);
    MAT = param_tbl[bid][Y & 0x0f];
    return Y ^ MAT;
}

/**
 * The tempering function.
 *
 * @param[in] V the output value should be tempered.
 * @param[in] T the tempering helper value.
 * @param[in] bid block id.
 * @return the tempered value.
 */
__device__ uint32_t temper(uint32_t V, uint32_t T, int bid) {
    uint32_t MAT;

    T ^= T >> 16;
    T ^= T >> 8;
    MAT = temper_tbl[bid][T & 0x0f];
    return V ^ MAT;
}

/**
 * The tempering and converting function.
 * By using the preset-ted table, converting to IEEE format
 * and tempering are done simultaneously.
 *
 * @param[in] V the output value should be tempered.
 * @param[in] T the tempering helper value.
 * @param[in] bid block id.
 * @return the tempered and converted value.
 */
__device__ uint32_t temper_single(uint32_t V, uint32_t T, int bid) {
    uint32_t MAT;
    uint32_t r;

    T ^= T >> 16;
    T ^= T >> 8;
    MAT = single_temper_tbl[bid][T & 0x0f];
    r = (V >> 9) ^ MAT;
    return r;
}

/**
 * Read the internal state vector from kernel I/O data, and
 * put them into shared memory.
 *
 * @param[out] status shared memory.
 * @param[in] d_status kernel I/O data
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_read(uint32_t status[LARGE_SIZE],
			    const mtgp32_kernel_status_t *d_status,
			    int bid,
			    int tid) {
    status[LARGE_SIZE - N + tid] = d_status[bid].status[tid];
    if (tid < N - THREAD_NUM) {
	status[LARGE_SIZE - N + THREAD_NUM + tid]
	    = d_status[bid].status[THREAD_NUM + tid];
    }
    __syncthreads();
}

/**
 * Read the internal state vector from shared memory, and
 * write them into kernel I/O data.
 *
 * @param[out] d_status kernel I/O data
 * @param[in] status shared memory.
 * @param[in] bid block id
 * @param[in] tid thread id
 */
__device__ void status_write(mtgp32_kernel_status_t *d_status,
			     const uint32_t status[LARGE_SIZE],
			     int bid,
			     int tid) {
    d_status[bid].status[tid] = status[LARGE_SIZE - N + tid];
    if (tid < N - THREAD_NUM) {
	d_status[bid].status[THREAD_NUM + tid]
	    = status[4 * THREAD_NUM - N + tid];
    }
    __syncthreads();
}

/**
 * kernel function.
 * This function generates 32-bit unsigned integers in d_data
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output
 * @param[in] size number of output data requested.
 */
__global__ void mtgp32_uint32_kernel(mtgp32_kernel_status_t* d_status,
				     uint32_t* d_data, int size) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t r;
    uint32_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {

#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[LARGE_SIZE - N + tid]:%08x\n",
		   status[LARGE_SIZE - N + tid]);
	    printf("status[LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[LARGE_SIZE - N + tid + 1]);
	    printf("status[LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[LARGE_SIZE - N + tid + pos]);
	    printf("sh1:%d\n", sh1_tbl[bid]);
	    printf("sh2:%d\n", sh2_tbl[bid]);
	    printf("mask:%08x\n", mask);
	    for (int j = 0; j < 16; j++) {
		printf("tbl[%d]:%08x\n", j, param_tbl[0][j]);
	    }
	}
#endif
	r = para_rec(status[LARGE_SIZE - N + tid],
		 status[LARGE_SIZE - N + tid + 1],
		 status[LARGE_SIZE - N + tid + pos],
		 bid);
	status[tid] = r;
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[tid]:%08x\n", status[tid]);
	}
#endif
	o = temper(r, status[LARGE_SIZE - N + tid + pos - 1], bid);
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("r:%08" PRIx32 "\n", r);
	}
#endif
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	r = para_rec(status[(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		     bid);
	status[tid + THREAD_NUM] = r;
	o = temper(r,
		   status[(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
		   bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	r = para_rec(status[2 * THREAD_NUM - N + tid],
		     status[2 * THREAD_NUM - N + tid + 1],
		     status[2 * THREAD_NUM - N + tid + pos],
		     bid);
	status[tid + 2 * THREAD_NUM] = r;
	o = temper(r, status[tid + pos - 1 + 2 * THREAD_NUM - N], bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * kernel function.
 * This function generates single precision floating point numbers in d_data.
 *
 * @param[in,out] d_status kernel I/O data
 * @param[out] d_data output. IEEE single precision format.
 * @param[in] size number of output data requested.
 */
__global__ void mtgp32_single_kernel(mtgp32_kernel_status_t* d_status,
				     uint32_t* d_data, int size)
{

    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t r;
    uint32_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {
	r = para_rec(status[LARGE_SIZE - N + tid],
		     status[LARGE_SIZE - N + tid + 1],
		     status[LARGE_SIZE - N + tid + pos],
		     bid);
	status[tid] = r;
	o = temper_single(r, status[LARGE_SIZE - N + tid + pos - 1], bid);
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	r = para_rec(status[(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		     status[(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		     bid);
	status[tid + THREAD_NUM] = r;
	o = temper_single(
	    r,
	    status[(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
	    bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	r = para_rec(status[2 * THREAD_NUM - N + tid],
		     status[2 * THREAD_NUM - N + tid + 1],
		     status[2 * THREAD_NUM - N + tid + pos],
		     bid);
	status[tid + 2 * THREAD_NUM] = r;
	o = temper_single(r,
			  status[tid + pos - 1 + 2 * THREAD_NUM - N],
			  bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * This function sets constants in device memory.
 * @param[in] params input, MTGP32 parameters.
 */
void make_constant(const mtgp32_params_fast_t params[]) {
    const int size1 = sizeof(uint32_t) * BLOCK_NUM;
    const int size2 = sizeof(uint32_t) * BLOCK_NUM * TBL_SIZE;
    uint32_t *h_pos_tbl;
    uint32_t *h_sh1_tbl;
    uint32_t *h_sh2_tbl;
    uint32_t *h_param_tbl;
    uint32_t *h_temper_tbl;
    uint32_t *h_single_temper_tbl;
#if 0
    uint32_t *h_mask;
#endif
    h_pos_tbl = (uint32_t *)malloc(size1);
    h_sh1_tbl = (uint32_t *)malloc(size1);
    h_sh2_tbl = (uint32_t *)malloc(size1);
    h_param_tbl = (uint32_t *)malloc(size2);
    h_temper_tbl = (uint32_t *)malloc(size2);
    h_single_temper_tbl = (uint32_t *)malloc(size2);
#if 0
    h_mask = (uint32_t *)malloc(sizeof(uint32_t));
#endif
    if (h_pos_tbl == NULL
	|| h_sh1_tbl == NULL
	|| h_sh2_tbl == NULL
	|| h_param_tbl == NULL
	|| h_temper_tbl == NULL
	|| h_single_temper_tbl == NULL
#if 0
	|| h_mask == NULL
#endif
	) {
	printf("failure in allocating host memory for constant table.\n");
	exit(1);
    }
#if 0
    h_mask = params[0].mask;
#endif
    for (int i = 0; i < BLOCK_NUM; i++) {
	h_pos_tbl[i] = params[i].pos;
	h_sh1_tbl[i] = params[i].sh1;
	h_sh2_tbl[i] = params[i].sh2;
	for (int j = 0; j < TBL_SIZE; j++) {
	    h_param_tbl[i * TBL_SIZE + j] = params[i].tbl[j];
	    h_temper_tbl[i * TBL_SIZE + j] = params[i].tmp_tbl[j];
	    h_single_temper_tbl[i * TBL_SIZE + j] = params[i].flt_tmp_tbl[j];
	}
    }
    // copy from malloc area only
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pos_tbl), h_pos_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh1_tbl), h_sh1_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh2_tbl), h_sh2_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(param_tbl), h_param_tbl, size2));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(temper_tbl), h_temper_tbl, size2));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(single_temper_tbl),
				      h_single_temper_tbl, size2));
#if 0
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&mask),
				      &h_mask, sizeof(uint32_t)));
#endif
    free(h_pos_tbl);
    free(h_sh1_tbl);
    free(h_sh2_tbl);
    free(h_param_tbl);
    free(h_temper_tbl);
    free(h_single_temper_tbl);
#if 0
    free(h_mask);
#endif
}

/**
 * This function initializes kernel I/O data.
 * @param[out] d_status output kernel I/O data.
 * @param[in] params MTGP32 parameters. needed for the initialization.
 */
void make_kernel_data(mtgp32_kernel_status_t *d_status,
		     mtgp32_params_fast_t params[]) {
    mtgp32_kernel_status_t* h_status = (mtgp32_kernel_status_t *) malloc(
	sizeof(mtgp32_kernel_status_t) * BLOCK_NUM);

    if (h_status == NULL) {
	printf("failure in allocating host memory for kernel I/O data.\n");
	exit(8);
    }
    for (int i = 0; i < BLOCK_NUM; i++) {
	mtgp32_init_state(&(h_status[i].status[0]), &params[i], i + 1);
    }
#if defined(DEBUG)
    printf("h_status[0].status[0]:%08"PRIx32"\n", h_status[0].status[0]);
    printf("h_status[0].status[1]:%08"PRIx32"\n", h_status[0].status[1]);
    printf("h_status[0].status[2]:%08"PRIx32"\n", h_status[0].status[2]);
    printf("h_status[0].status[3]:%08"PRIx32"\n", h_status[0].status[3]);
#endif
    CUDA_SAFE_CALL(hipMemcpy(d_status,
			      h_status,
			      sizeof(mtgp32_kernel_status_t) * BLOCK_NUM,
			      hipMemcpyHostToDevice));
    free(h_status);
}

/**
 * This function is used to compare the outputs with C program's.
 * @param[in] array data to be printed.
 * @param[in] size size of array.
 * @param[in] block number of blocks.
 */
void print_float_array(const float array[], int size, int block) {
    int b = size / block;

    for (int j = 0; j < 5; j += 5) {
	printf("%.10f %.10f %.10f %.10f %.10f\n",
	       array[j], array[j + 1],
	       array[j + 2], array[j + 3], array[j + 4]);
    }
    for (int i = 1; i < block; i++) {
	for (int j = -5; j < 5; j += 5) {
	    printf("%.10f %.10f %.10f %.10f %.10f\n",
		   array[b * i + j],
		   array[b * i + j + 1],
		   array[b * i + j + 2],
		   array[b * i + j + 3],
		   array[b * i + j + 4]);
	}
    }
    for (int j = -5; j < 0; j += 5) {
	printf("%.10f %.10f %.10f %.10f %.10f\n",
	       array[size + j],
	       array[size + j + 1],
	       array[size + j + 2],
	       array[size + j + 3],
	       array[size + j + 4]);
    }
}

/**
 * This function is used to compare the outputs with C program's.
 * @param[in] array data to be printed.
 * @param[in] size size of array.
 * @param[in] block number of blocks.
 */
void print_uint32_array(uint32_t array[], int size, int block) {
    int b = size / block;

    for (int j = 0; j < 5; j += 5) {
	printf("%10" PRIu32 " %10" PRIu32 " %10" PRIu32
	       " %10" PRIu32 " %10" PRIu32 "\n",
	       array[j], array[j + 1],
	       array[j + 2], array[j + 3], array[j + 4]);
    }
    for (int i = 1; i < block; i++) {
	for (int j = -5; j < 5; j += 5) {
	    printf("%10" PRIu32 " %10" PRIu32 " %10" PRIu32
		   " %10" PRIu32 " %10" PRIu32 "\n",
		   array[b * i + j],
		   array[b * i + j + 1],
		   array[b * i + j + 2],
		   array[b * i + j + 3],
		   array[b * i + j + 4]);
	}
    }
    for (int j = -5; j < 0; j += 5) {
	printf("%10" PRIu32 " %10" PRIu32 " %10" PRIu32
	       " %10" PRIu32 " %10" PRIu32 "\n",
	       array[size + j],
	       array[size + j + 1],
	       array[size + j + 2],
	       array[size + j + 3],
	       array[size + j + 4]);
    }
}
/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
void make_uint32_random(mtgp32_kernel_status_t* d_status, int num_data) {
    uint32_t* d_data;
    unsigned int timer = 0;
    uint32_t* h_data;
    hipError_t e;
    float gputime;

    printf("generating 32-bit unsigned random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint32_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (uint32_t *) malloc(sizeof(uint32_t) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp32_uint32_kernel<<< BLOCK_NUM, THREAD_NUM>>>(
	d_status, d_data, num_data / BLOCK_NUM);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint32_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_uint32_array(h_data, num_data, BLOCK_NUM);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param[in] d_status kernel I/O data.
 * @param[in] num_data number of data to be generated.
 */
void make_single_random(mtgp32_kernel_status_t* d_status, int num_data) {
    uint32_t* d_data;
    unsigned int timer = 0;
    float* h_data;
    hipError_t e;
    float gputime;

    printf("generating single precision floating point random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint32_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (float *) malloc(sizeof(float) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp32_single_kernel<<< BLOCK_NUM, THREAD_NUM >>>(
	d_status, d_data, num_data / BLOCK_NUM);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint32_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_float_array(h_data, num_data, BLOCK_NUM);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

int main(int argc, char** argv)
{
    // LARGE_SIZE is a multiple of 16
    int num_data = 10000000;
    int num_unit = LARGE_SIZE * BLOCK_NUM;
    int r;
    mtgp32_kernel_status_t* d_status;

    CUT_DEVICE_INIT(argc, argv);
    CUDA_SAFE_CALL(hipMalloc((void**)&d_status,
			      sizeof(mtgp32_kernel_status_t) * BLOCK_NUM));
    if (argc >= 2) {
	errno = 0;
	num_data = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_output\n", argv[0]);
	    return 1;
	}
    } else {
	printf("%s number_of_output\n", argv[0]);
	return 1;
    }
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    make_constant(mtgp32_params_fast_23209);
    make_kernel_data(d_status, mtgp32_params_fast_23209);
    make_uint32_random(d_status, num_data);
    make_single_random(d_status, num_data);

    //finalize
    CUDA_SAFE_CALL(hipFree(d_status));
    CUT_EXIT(argc, argv);
}
